#include "hip/hip_runtime.h"
#include "bonsai.h"
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include "vector_math.h"

#include <GL/glew.h>
#include <cuda_gl_interop.h>

// calculate eye-space depth for each particle
KERNEL_DECLARE(calcDepthKernel)(float4 *pos, float *depth, int *indices, float4 modelViewZ, int numParticles)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i >= numParticles) return;

	float4 p = pos[i];
	float z = dot(make_float4(p.x, p.y, p.z, 1.0f), modelViewZ);
	
	depth[i] = z;
	indices[i] = i;
}

void thrustSort(float* keys, int* values, int count)
{
    thrust::device_ptr<float> dkeys(keys);
    thrust::device_ptr<int> dvalues(values);
    thrust::sort_by_key(dkeys, dkeys + count, dvalues);
}

extern "C"
void initCUDA()
{
    cudaGLSetGLDevice(0);
}

extern "C"
void depthSortCUDA(float4 *pos, float *depth, int *indices, float4 modelViewZ, int numParticles)
{
	int numThreads = 256;
	int numBlocks = (numParticles + numThreads - 1) / numThreads;
    calcDepthKernel<<< numBlocks, numThreads >>>(pos, depth, indices, modelViewZ, numParticles);

	thrustSort(depth, indices, numParticles);
}