#include "hip/hip_runtime.h"
// //#include "/home/jbedorf/papers/GBPZ2010/codes/jb/build_tree/CUDA/support_kernels.cu"
#include "support_kernels.cu"
#include <stdio.h>
#include "octree.h"

#include "../profiling/bonsai_timing.h"
PROF_MODULE(build_tree);

//////////////////////////////
//////////////////////////////
//////////////////////////////
#define LEVEL_MIN 3

extern "C" __global__ void boundaryReduction(const int n_particles,
                                            real4      *positions,
                                            float3     *output_min,
                                            float3     *output_max)
{
  CUXTIMER("boundaryReduction");
  const uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  const uint tid = threadIdx.x;
  //const uint idx = bid * blockDim.x + tid;

  volatile __shared__ float3 shmem[512];
  float3 r_min = make_float3(+1e10f, +1e10f, +1e10f);
  float3 r_max = make_float3(-1e10f, -1e10f, -1e10f);

  volatile float3 *sh_rmin = (float3*)&shmem [ 0];
  volatile float3 *sh_rmax = (float3*)&shmem[256];
  sh_rmin[tid].x = r_min.x; sh_rmin[tid].y = r_min.y; sh_rmin[tid].z = r_min.z;
  sh_rmax[tid].x = r_max.x; sh_rmax[tid].y = r_max.y; sh_rmax[tid].z = r_max.z;

  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  const int blockSize   = blockDim.x;
//   unsigned int tid      = threadIdx.x;
  unsigned int i        = blockIdx.x*(blockSize*2) + threadIdx.x;
  unsigned int gridSize = blockSize*2*gridDim.x;

  real4 pos;
  // we reduce multiple elements per thread.  The number is determined by the
  // number of active thread blocks (via gridSize).  More blocks will result
  // in a larger gridSize and therefore fewer elements per thread
  //based on reduce6 example
  while (i < n_particles) {
    if (i             < n_particles)
    {
      pos = positions[i];
      r_min.x = fminf(pos.x, r_min.x);
      r_min.y = fminf(pos.y, r_min.y);
      r_min.z = fminf(pos.z, r_min.z);
      r_max.x = fmaxf(pos.x, r_max.x);
      r_max.y = fmaxf(pos.y, r_max.y);
      r_max.z = fmaxf(pos.z, r_max.z);
    }
    if (i + blockSize < n_particles)
    {
      pos = positions[i + blockSize];
      r_min.x = fminf(pos.x, r_min.x);
      r_min.y = fminf(pos.y, r_min.y);
      r_min.z = fminf(pos.z, r_min.z);
      r_max.x = fmaxf(pos.x, r_max.x);
      r_max.y = fmaxf(pos.y, r_max.y);
      r_max.z = fmaxf(pos.z, r_max.z);
    }
    i += gridSize;
  }

  sh_rmin[tid].x = r_min.x; sh_rmin[tid].y = r_min.y; sh_rmin[tid].z = r_min.z;
  sh_rmax[tid].x = r_max.x; sh_rmax[tid].y = r_max.y; sh_rmax[tid].z = r_max.z;

  __syncthreads();
  // do reduction in shared mem  
  if(blockDim.x >= 512) if (tid < 256) {sh_MinMax(tid, tid + 256, &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();
  if(blockDim.x >= 256) if (tid < 128) {sh_MinMax(tid, tid + 128, &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();
  if(blockDim.x >= 128) if (tid < 64)  {sh_MinMax(tid, tid + 64,  &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();

  if (tid < 32) 
  {
    sh_MinMax(tid, tid + 32, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid + 16, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  8, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  4, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  2, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  1, &r_min, &r_max, sh_rmin,sh_rmax);
  }

  // write result for this block to global mem
  if (tid == 0)
  {
    //Compiler doesnt allow: volatile float3 = float3
    output_min[bid].x = sh_rmin[0].x; output_min[bid].y = sh_rmin[0].y; output_min[bid].z = sh_rmin[0].z;
    output_max[bid].x = sh_rmax[0].x; output_max[bid].y = sh_rmax[0].y; output_max[bid].z = sh_rmax[0].z;
  }

}


//Get the domain size, by taking into account the group size
extern "C" __global__ void boundaryReductionGroups(const int n_groups,
                                                   real4      *positions,
                                                   real4      *sizes,
                                                   float3     *output_min,
                                                   float3     *output_max)
{
  CUXTIMER("boundaryReductionGroups");
  const uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  const uint tid = threadIdx.x;
  //const uint idx = bid * blockDim.x + tid;

  volatile __shared__ float3 shmem[512];
  float3 r_min = make_float3(+1e10f, +1e10f, +1e10f);
  float3 r_max = make_float3(-1e10f, -1e10f, -1e10f);

  volatile float3 *sh_rmin = (float3*)&shmem [ 0];
  volatile float3 *sh_rmax = (float3*)&shmem[256];
  sh_rmin[tid].x = r_min.x; sh_rmin[tid].y = r_min.y; sh_rmin[tid].z = r_min.z;
  sh_rmax[tid].x = r_max.x; sh_rmax[tid].y = r_max.y; sh_rmax[tid].z = r_max.z;

  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  const int blockSize   = blockDim.x;
//   unsigned int tid      = threadIdx.x;
  unsigned int i        = blockIdx.x*(blockSize*2) + threadIdx.x;
  unsigned int gridSize = blockSize*2*gridDim.x;

  real4 pos;
  real4 size;
  // we reduce multiple elements per thread.  The number is determined by the
  // number of active thread blocks (via gridSize).  More blocks will result
  // in a larger gridSize and therefore fewer elements per thread
  //based on reduce6 example
  while (i < n_groups) {
    if (i             < n_groups)
    {
      pos = positions[i];
      size = sizes[i];
      r_min.x = fminf(pos.x-size.x, r_min.x);
      r_min.y = fminf(pos.y-size.y, r_min.y);
      r_min.z = fminf(pos.z-size.z, r_min.z);
      r_max.x = fmaxf(pos.x+size.x, r_max.x);
      r_max.y = fmaxf(pos.y+size.y, r_max.y);
      r_max.z = fmaxf(pos.z+size.z, r_max.z);
    }
    if (i + blockSize < n_groups)
    {
      pos = positions[i + blockSize];
      size = sizes[i + blockSize];
      r_min.x = fminf(pos.x-size.x, r_min.x);
      r_min.y = fminf(pos.y-size.y, r_min.y);
      r_min.z = fminf(pos.z-size.z, r_min.z);
      r_max.x = fmaxf(pos.x+size.x, r_max.x);
      r_max.y = fmaxf(pos.y+size.y, r_max.y);
      r_max.z = fmaxf(pos.z+size.z, r_max.z);
    }
    i += gridSize;
  }

  sh_rmin[tid].x = r_min.x; sh_rmin[tid].y = r_min.y; sh_rmin[tid].z = r_min.z;
  sh_rmax[tid].x = r_max.x; sh_rmax[tid].y = r_max.y; sh_rmax[tid].z = r_max.z;

  __syncthreads();
  // do reduction in shared mem  
  if(blockDim.x >= 512) if (tid < 256) {sh_MinMax(tid, tid + 256, &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();
  if(blockDim.x >= 256) if (tid < 128) {sh_MinMax(tid, tid + 128, &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();
  if(blockDim.x >= 128) if (tid < 64)  {sh_MinMax(tid, tid + 64,  &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();

  if (tid < 32) 
  {
    sh_MinMax(tid, tid + 32, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid + 16, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  8, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  4, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  2, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  1, &r_min, &r_max, sh_rmin,sh_rmax);
  }

  // write result for this block to global mem
  if (tid == 0)
  {
    //Compiler doesnt allow: volatile float3 = float3
    output_min[bid].x = sh_rmin[0].x; output_min[bid].y = sh_rmin[0].y; output_min[bid].z = sh_rmin[0].z;
    output_max[bid].x = sh_rmax[0].x; output_max[bid].y = sh_rmax[0].y; output_max[bid].z = sh_rmax[0].z;
  }

}

//#define EXACT_KEY

extern "C" __global__ void cl_build_key_list(uint4  *body_key,
                                            real4  *body_pos,
                                            int   n_bodies,
                                            real4  corner) {
  
  CUXTIMER("cl_build_key_list");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;
  
  if (id > n_bodies) return;

  real4 pos = body_pos[id];

  int4 crd;
  
  real domain_fac = corner.w;
  
  #ifndef EXACT_KEY
    crd.x = (int)roundf(__fdividef((pos.x - corner.x), domain_fac));
    crd.y = (int)roundf(__fdividef((pos.y - corner.y) , domain_fac));
    crd.z = (int)roundf(__fdividef((pos.z - corner.z) , domain_fac));
  #else            
    crd.x = (int)((pos.x - corner.x) / domain_fac);
    crd.y = (int)((pos.y - corner.y) / domain_fac);
    crd.z = (int)((pos.z - corner.z) / domain_fac);
  #endif

   uint4 key = get_key(crd);


//   if (id == n_bodies) key = make_uint4(0xFFFFFFFF, 0xFFFFFFFF, 0, 0);
  if (id == n_bodies) key = make_uint4(0xFFFFFFFF, 0xFFFFFFFF, 0, 0);

  key.w = id;

  body_key[id] = key;

}

    
  

extern "C" __global__ void cl_build_valid_list(int n_bodies,
                                               int level,
                                               uint4  *body_key,
                                               uint *valid_list,
                                               const uint *workToDo) {
  if (0 == *workToDo) return;
//                                                uint2 *test_key_data) {
  CUXTIMER("cl_build_valid_list");
  const uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  const uint tid = threadIdx.x;
  const uint id  = bid * blockDim.x + tid;
  const uint4 key_F = {0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF};
  
  if (id >= n_bodies) return;   // >=   since the last particle is extra boudnary particle
  
  uint4 mask = get_mask(level);
  mask.x = mask.x | ((uint)1 << 30) | ((uint)1 << 31);

  uint4 key_m;
  uint4 key_c    = body_key[id];
  uint4 key_p;

  if (id == 0)
  {
    key_m = key_F;
  }
  else
  {
    key_m = body_key[id-1];
  }

  if((id+1) <  n_bodies) //The last particle gets a different key to compare with
  {
    key_p = body_key[id+1];
  }
  else
    key_p = make_uint4(0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF);


  int valid0 = 0;
  int valid1 = 0;

  if (cmp_uint4(key_c, key_F) != 0) {
    key_c.x = key_c.x & mask.x;
    key_c.y = key_c.y & mask.y;
    key_c.z = key_c.z & mask.z;

    key_p.x = key_p.x & mask.x;
    key_p.y = key_p.y & mask.y;
    key_p.z = key_p.z & mask.z;

    key_m.x = key_m.x & mask.x;
    key_m.y = key_m.y & mask.y;
    key_m.z = key_m.z & mask.z;

    valid0 = abs(cmp_uint4(key_c, key_m));
    valid1 = abs(cmp_uint4(key_c, key_p));
  }

   valid_list[id*2]   = id | ((valid0) << 31);
   valid_list[id*2+1] = id | ((valid1) << 31);

}


//////////////////////////////
//////////////////////////////
//////////////////////////////
__device__ uint retirementCountBuildNodes = 0;

extern "C" __global__ void cl_build_nodes(uint level,
                             uint  *compact_list_len,
                             uint  *level_offset,
                             uint  *last_level,
                             uint2 *level_list,
                             uint  *compact_list,
                             uint4 *bodies_key,
                             uint4 *node_key,
                             uint  *n_children,
                             uint2 *node_bodies){

  CUXTIMER("cl_build_nodes");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;

  uint n = (*compact_list_len)/2;
  uint offset = *level_offset;

  for (; id < n; id += gridDim.x * gridDim.y * blockDim.x)
  {
    uint  bi   = compact_list[id*2];
    uint  bj   = compact_list[id*2+1] + 1;
  
    uint4 key  = bodies_key[bi];
    uint4 mask = get_mask(level);
    key = make_uint4(key.x & mask.x, key.y & mask.y, key.z & mask.z, 0); 

    node_bodies[offset+id] = make_uint2(bi | (level << BITLEVELS), bj);
    node_key   [offset+id] = key;
    n_children [offset+id] = 0;
  
    if ((int)level > (int)(LEVEL_MIN - 1)) 
      if (bj - bi <= NLEAF)                            //Leaf can only have NLEAF particles, if its more there will be a split
        for (int i = bi; i < bj; i++)
          bodies_key[i] = make_uint4(0xFFFFFFFF,0xFFFFFFFF,0xFFFFFFFF,0xFFFFFFFF); //sets the key to FF to indicate the body is used
  }

  //
  // PHASE 2: Last block updates level list and offset
  //

  int numBlocks = gridDim.x * gridDim.y;
  if (numBlocks > 1)
  {
    __shared__ bool amLast;

    // Thread 0 takes a ticket
    if( tid==0 )
    {
      unsigned int ticket = atomicInc(&retirementCountBuildNodes, numBlocks);
      // If the ticket ID is equal to the number of blocks, we are the last block!
      amLast = (ticket == numBlocks-1);
    }
    __syncthreads();

    // The last block sums the results of all other blocks
    if( amLast && tid == 0)
    {           
      level_list[level] = (n > 0) ? make_uint2(offset, offset + n) : make_uint2(0, 0);
      *level_offset = offset + n;

      if ((level > 0) && (n <= 0) && (level_list[level - 1].x > 0))
        *last_level = level;

      // reset retirement count so that next run succeeds
      retirementCountBuildNodes = 0; 
    }
  }
}

void build_tree_node_levels(octree &tree, 
                            my_dev::dev_mem<uint>  &validList,
                            my_dev::dev_mem<uint>  &compactList,
                            my_dev::dev_mem<uint>  &levelOffset,
                            my_dev::dev_mem<uint>  &maxLevel)
{
   // set devMemCountsx to 1 because it is used to early out when it hits zero
  tree.devMemCountsx[0] = 1;
  tree.devMemCountsx.h2d(1);

  dim3 grid, block;

  //int nodeSum = 0;
  for (uint level = 0; level < MAXLEVELS; level++) {
    // mark bodies to be combined into nodes
    //Calculate dynamic
    int ng = (tree.localTree.n) / 128 + 1;
    grid.x = (int)sqrt((double)ng);
    grid.y = (ng -1)/grid.x +  1; 
    grid.z = 1;
    block.x = 128; block.y = block.z = 1;

    cl_build_valid_list<<<grid, block>>>(tree.localTree.n, 
                                         level, 
                                         tree.localTree.bodies_key.raw_p(),
                                         validList.raw_p(), 
                                         tree.devMemCountsx.raw_p());
      
    //gpuCompact to get number of created nodes    
    tree.gpuCompact(*tree.getDevContext(), validList, compactList, tree.localTree.n*2, 0);
                   
    // assemble nodes   
    grid.x = (120*32)/128; grid.y = 4; grid.z = 1;
    block.x = 128; block.y = 1; block.z = 1;

    cl_build_nodes<<<grid, block>>>(level, 
                                    tree.devMemCountsx.raw_p(), 
                                    levelOffset.raw_p(), 
                                    maxLevel.raw_p(),
                                    tree.localTree.level_list.raw_p(), 
                                    compactList.raw_p(),
                                    tree.localTree.bodies_key.raw_p(),
                                    tree.localTree.node_key.raw_p(),
                                    tree.localTree.n_children.raw_p(),
                                    tree.localTree.node_bodies.raw_p());
  } //end for lvl

  // reset counts to 1 so next compact proceeds...
  tree.devMemCountsx[0] = 1;
  tree.devMemCountsx.h2d(1); 
}

//////////////////////////////
//////////////////////////////
//////////////////////////////


extern "C" __global__ void cl_link_tree(int n_nodes,
                            uint *n_children,
                            uint2 *node_bodies,
                            real4 *bodies_pos,
                            real4 corner,
                            uint2 *level_list,           //TODO could make this constant if it proves usefull
                            uint* valid_list,
                            uint4 *node_keys,
                            uint4 *bodies_key) {

  CUXTIMER("cl_link_tree");
  const uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  const uint tid = threadIdx.x;
  uint id  = bid * blockDim.x + tid;
  
  if (id >= n_nodes) return;

  uint2 bij  = node_bodies[id];
  uint level = (bij.x &  LEVELMASK) >> BITLEVELS;
  uint bi    =  bij.x & ILEVELMASK;
  uint bj    =  bij.y;

  real4 pos  = bodies_pos[bi];
  int4 crd;
  real domain_fac = corner.w;

  #ifndef EXACT_KEY
    crd.x = (int)roundf(__fdividef((pos.x - corner.x), domain_fac));
    crd.y = (int)roundf(__fdividef((pos.y - corner.y) , domain_fac));
    crd.z = (int)roundf(__fdividef((pos.z - corner.z) , domain_fac));
  #else            
    crd.x = (int)((pos.x - corner.x) / domain_fac);
    crd.y = (int)((pos.y - corner.y) / domain_fac);
    crd.z = (int)((pos.z - corner.z) / domain_fac);
  #endif


  uint4 key = get_key(crd);


  /********* accumulate children *****/
  
  uint4 mask = get_mask(level - 1);
  key = make_uint4(key.x & mask.x, key.y & mask.y,  key.z & mask.z, 0); 

  uint2 cij;

  
  if(id > 0)
    cij = level_list[level-1];

  int ci;
  //Jeroen, modified this since we dont use textures in find_key,
  //the function will fail because out of bound memory access when id==0
  if(id > 0)
    ci = find_key(key, cij, node_keys);
  else
    ci = 0;

  //ci now points to the node that is the parent, was used in previous group method
  //parent_id_list[id] = ci;

  mask = get_imask(mask);
  key = make_uint4(key.x | mask.x, key.y | mask.y, key.z | mask.z, 0);
  if (id > 0)   
    atomicAdd(&n_children[ci], (1 << 28));

  key = get_key(crd);
  mask = get_mask(level);
  key = make_uint4(key.x & mask.x, key.y & mask.y, key.z & mask.z, 0); 

  /********* store the 1st child *****/

  cij = level_list[level+1];
  int cj = -1;

  cj = find_key(key, cij, node_keys);

  atomicOr(&n_children[id], cj); //Atomic since multiple threads can work on this

  uint valid =  id | (uint)(0 << 31); 

  
  if ((int)level > (int)(LEVEL_MIN - 1)) 
    if ((bj - bi) <= NLEAF)    
      valid = id | (uint)(1 << 31);   //Distinguish leaves and nodes

 valid_list[id] = valid; //If valid its a leaf otherwise a node
}

//Determines which level of node starts at which offset
extern "C" __global__ void build_level_list(const int n_nodes,
                                            const int n_leafs,
                                            uint *leafsIdxs,
                                            uint2 *node_bodies,                                      
                                            uint* valid_list)
{
  CUXTIMER("build_level_list");
  const uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  const uint tid = threadIdx.x;
  const uint id  = bid * blockDim.x + tid;
  
  if (id >= n_nodes-n_leafs) return;

  const int nodeID = leafsIdxs[id+n_leafs];   //Get the idx into the node_bodies array

  int level_c, level_m, level_p;


  uint2 bij   = node_bodies[leafsIdxs[id+n_leafs]];    //current non-leaf
  level_c     = (bij.x &  LEVELMASK) >> BITLEVELS;

  if((id+1) < (n_nodes-n_leafs))        //The last node gets a default lvl
  {
    bij         = node_bodies[leafsIdxs[id+1+n_leafs]]; //next non-leaf
    level_p     = (bij.x &  LEVELMASK) >> BITLEVELS;
  }
  else
    level_p     = MAXLEVELS+5;  //Last is always an end

  //Compare level with the node before and node after
  if(nodeID == 0)
  {
    level_m = -1;    
  }
  else
  {
    bij         = node_bodies[ leafsIdxs[id-1+n_leafs]]; //Get info of previous non-leaf node
    level_m     =  (bij.x &  LEVELMASK) >> BITLEVELS;   
  }

  int valid0 = 0;
  int valid1 = 0;

  valid0 = (level_c != level_m) << 31 | (id+n_leafs);
  valid1 = (level_c != level_p) << 31 | (id+n_leafs);

  valid_list[id*2]   = valid0;
  valid_list[id*2+1] = valid1;

} //end build_level_list


//Finds nodes/leafs that will become groups
//After executions valid_list contains the 
//valid nodes/leafs that form groups
extern "C" __global__ void build_group_list2(int    n_particles,
                                             uint  *validList,
                                             real4  *bodies_pos,
                                             const float DIST,
                                             int   *node_level_list,
                                             int   treeDepth)
{
  CUXTIMER("build_group_list2");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint idx = bid * blockDim.x + tid;

  __shared__ int shmem[128];

  //Compact the node_level_list
  if(bid == 0)
  {
    if(threadIdx.x < (MAXLEVELS*2))
    {
      shmem[threadIdx.x] = node_level_list[threadIdx.x];
    }

    __syncthreads(); //Can most likely do without since its one warp

    //Only selection writes
    if(threadIdx.x < MAXLEVELS)
    {
      node_level_list[threadIdx.x]  = shmem[threadIdx.x*2];
      if(threadIdx.x == treeDepth-1)
          node_level_list[threadIdx.x] = shmem[threadIdx.x*2-1]+1;
    }
  }//if bid == 0
  //end compact node level list

  //Note that we do not include the final particle
  //Since there is no reason to check it
  if (idx >= n_particles) return;

  //Get the current 
  float4 curPos, nexPos, prevPos;

  curPos  =  bodies_pos[idx];

  //Have to check the first and last to prevent out of bound access
  if(idx+1 == n_particles)
    nexPos  =  curPos;
  else
    nexPos = bodies_pos[idx+1];

  if(idx == 0)
    prevPos = curPos;
  else
    prevPos =  bodies_pos[idx-1];

  //Compute geometrical distance
  float dsPlus = ((curPos.x-nexPos.x)*(curPos.x-nexPos.x)) + 
                 ((curPos.y-nexPos.y)*(curPos.y-nexPos.y)) + 
                 ((curPos.z-nexPos.z)*(curPos.z-nexPos.z));

  float dsMin = ((curPos.x-prevPos.x)*(curPos.x-prevPos.x)) + 
                ((curPos.y-prevPos.y)*(curPos.y-prevPos.y)) + 
                ((curPos.z-prevPos.z)*(curPos.z-prevPos.z));

  //Multiples of the preferred group size are _always_ valid
  int validStart = ((idx     % NCRIT) == 0);
  int validEnd   = (((idx+1) % NCRIT) == 0);

  //The extra possible split(s) if the distance between two particles is too large
  if(dsPlus > DIST) validEnd     = 1;
  if(dsMin  > DIST) validStart   = 1;
  
  //Last particle is always the end, n_particles dont have to be a multiple of NCRIT
  //so this is required
  if(idx+1 == n_particles) validEnd = 1;

  //Set valid
  validList[2*idx + 0] = (idx)   | (uint)(validStart << 31);
  validList[2*idx + 1] = (idx+1) | (uint)(validEnd   << 31);    
}

 
//Store per particle the group id it belongs to
//and the start and end particle number of the groups  
extern "C" __global__ void store_group_list(int    n_particles,
                                            int n_groups,
                                            uint  *validList,
                                            uint  *body2group_list,
                                            uint2 *group_list)
{
  CUXTIMER("store_group_list");
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
//   uint idx = bid * blockDim.x + tid;
  
  if(bid >= n_groups) return;

  int start = validList[2*bid];
  int end   = validList[2*bid+1];

  if((start + tid) < end)
  {
    body2group_list[start + tid] = bid;
  }

  if(tid == 0)
  {
     group_list[bid] = make_uint2(start,end);
  }
}

//////////// Functions specific for dust //////////////////

extern "C" __global__ void define_dust_groups(int    n_particles,
					      real4  *dust_pos,
                                              uint  *validList)
{
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
  uint idx = bid * blockDim.x + tid;


  //Note that we do not include the final particle
  //Since there is no reason to check it
  if (idx >= n_particles) return;

 
  //Multiples of the preferred group size are _always_ valid
  int validStart = ((idx     % NCRIT) == 0);
  int validEnd   = (((idx+1) % NCRIT) == 0);


  //Get the current 
  float4 curPos, nexPos, prevPos;

  curPos  =  dust_pos[idx];

  //Have to check the first and last to prevent out of bound access
  if(idx+1 == n_particles)
    nexPos  =  curPos;
  else
    nexPos = dust_pos[idx+1];

  if(idx == 0)
    prevPos = curPos;
  else
    prevPos =  dust_pos[idx-1];

  //Compute geometrical distance
  float dsPlus = ((curPos.x-nexPos.x)*(curPos.x-nexPos.x)) + 
                 ((curPos.y-nexPos.y)*(curPos.y-nexPos.y)) + 
                 ((curPos.z-nexPos.z)*(curPos.z-nexPos.z));

  float dsMin = ((curPos.x-prevPos.x)*(curPos.x-prevPos.x)) + 
                ((curPos.y-prevPos.y)*(curPos.y-prevPos.y)) + 
                ((curPos.z-prevPos.z)*(curPos.z-prevPos.z));


  float DIST = 100;
  //The extra possible split(s) if the distance between two particles is too large
  if(dsPlus > DIST) validEnd     = 1;
  if(dsMin  > DIST) validStart   = 1;


  //Last particle is always the end, n_particles dont have to be a multiple of NCRIT
  //so this is required
  if(idx+1 == n_particles) validEnd = 1;

  //Set valid
  if(validStart)
    validList[2*idx + 0] = (idx)   | (uint)(validStart << 31);
  if(validEnd)
    validList[2*idx + 1] = (idx) | (uint)(validEnd   << 31);    
}

//JB: This one is slightly different from the store_group_list
//since  in my infinite wisdom I decided to make the comparisons
//slightly different when making the new define_dust_groups
extern "C" __global__ void store_dust_groups(int    n_groups,
                                            uint  *validList,
                                            uint  *body2group_list,
                                            uint2 *group_list,
                                            uint  *activeDustGroups)
{
  uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  uint tid = threadIdx.x;
//   uint idx = bid * blockDim.x + tid;
  
  if(bid >= n_groups) return;

  int start = validList[2*bid];
  int end   = validList[2*bid+1];

  if((start + tid) <= end)
  {
     body2group_list[start + tid] = bid;
  }

  if(tid == 0)
  {
     group_list[bid] = (uint2) make_uint2(start,end+1);
     activeDustGroups[bid] = bid;
  }
}

//This function stores the predicted position and velocity
//in the original array. This is used since it reduces
//memory storage and memory reorders after sorting 
//It is slightly less accurate and therefore not used 
//for the real bodies. In the correct function we compute back
extern "C" __global__ void predict_dust_particles(const int n_bodies,
                                                  float tc,
                                                  float tp,
                                                  real4 *pos,
                                                  real4 *vel,
                                                  real4 *acc,
                                                  uint  *body2grouplist,
                                                  uint  *valid_list){                                          
  const uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  const uint tid = threadIdx.x;
  const uint idx = bid * blockDim.x + tid;

  if (idx >= n_bodies) return;

  float4 p = pos [idx];
  float4 v = vel [idx];
  float4 a = acc [idx];

  float dt_cb  = tc - tp;

  p.x += v.x*dt_cb + a.x*dt_cb*dt_cb*0.5f;
  p.y += v.y*dt_cb + a.y*dt_cb*dt_cb*0.5f;
  p.z += v.z*dt_cb + a.z*dt_cb*dt_cb*0.5f;
  
  v.x += a.x*dt_cb;
  v.y += a.y*dt_cb;
  v.z += a.z*dt_cb;

  pos[idx] = p;
  vel[idx] = v;

  //This is needed to retain compatability with the original 
  //approximate gravity function
  int grpID = body2grouplist[idx];
  valid_list[grpID] = grpID; 
}



extern "C" __global__ void correct_dust_particles(const int n_bodies,
                                                  float dt_cb,
                                                  uint   *active_list,
                                                  real4 *vel,
                                                  real4 *acc0,
                                                  real4 *acc1) {
  const int bid =  blockIdx.y *  gridDim.x +  blockIdx.x;
  const int tid =  threadIdx.y * blockDim.x + threadIdx.x;
  const int dim =  blockDim.x * blockDim.y;

  int idx = bid * dim + tid;
  if (idx >= n_bodies) return;

  //Check if particle is set to active during approx grav
  #ifdef DO_BLOCK_TIMESTEP
    if (active_list[idx] != 1) return;
  #endif

  float4 a0 = acc0[idx];
  float4 a1 = acc1[idx];
  float4  v = vel[idx];

  //Correct the velocity
  dt_cb *= 0.5f;
  v.x += (a1.x - a0.x)*dt_cb;
  v.y += (a1.y - a0.y)*dt_cb;
  v.z += (a1.z - a0.z)*dt_cb;

  //Store the corrected velocity, accelaration and the new time step info
  vel     [idx] = v;
  acc0    [idx] = a1;
}

