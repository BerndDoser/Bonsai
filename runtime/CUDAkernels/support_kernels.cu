#include "hip/hip_runtime.h"
//Definitions
#ifndef _SUPPORT_KERNELS_
#define _SUPPORT_KERNELS_

#include "node_specs.h"

#include <stdio.h>

#ifdef USE_THRUST
  #include <thrust/device_ptr.h>
  #include <thrust/copy.h>
  #include <thrust/sort.h>
  #include <thrust/gather.h>
  #include <thrust/device_vector.h> 
  #include <thrust/iterator/transform_iterator.h>

  #include "../include/my_cuda_rt.h"

  extern "C" void thrust_sort_32b(my_dev::context &devContext, 
                      my_dev::dev_mem<uint> &srcKeys,     my_dev::dev_mem<uint> &srcValues,
                      my_dev::dev_mem<int>  &keysOutput,  my_dev::dev_mem<uint> &keysAPing,
                      my_dev::dev_mem<uint> &valuesOutput,my_dev::dev_mem<uint> &valuesAPing,
                      int N, int numberOfBits)
  {

      // wrap raw pointer with a device_ptr 
      thrust::device_ptr<uint> key_dev_ptr = thrust::device_pointer_cast(srcKeys.raw_p());
      thrust::device_ptr<uint> val_dev_ptr = thrust::device_pointer_cast(srcValues.raw_p());
      
      thrust::stable_sort_by_key(key_dev_ptr, key_dev_ptr + N, val_dev_ptr);
      
      valuesOutput.copy(srcValues, N);
  }  

  template <int keyIdx>
  struct ExtractBits : public thrust::unary_function<uint4, uint>
  {
    __host__ __device__ 
    uint operator()(uint4 key) const {
      if(keyIdx == 0)
        return key.x;
      else if(keyIdx == 1)
        return key.y;
      else 
        return key.z;
    }
  };
  
  template <int keyIdx, typename KeyPtr, typename PermutationPtr, typename ExtractedPtr>
  void update_permutation(KeyPtr& keys, PermutationPtr& permutation, ExtractedPtr& temp, int N)
  {
    // permute the keys with the current reordering
    thrust::gather(permutation, permutation + N, 
                   thrust::make_transform_iterator(keys, ExtractBits<keyIdx>()), temp);

    // stable_sort the permuted keys and update the permutation
    thrust::stable_sort_by_key(temp, temp + N, permutation);
  }


  template <typename KeyPtr, typename PermutationPtr, typename OutputPtr>
  void apply_permutation(KeyPtr& keys, PermutationPtr& permutation, OutputPtr& out, int N)
  {
    // permute the keys into out vector
    thrust::gather(permutation, permutation + N, keys, out);
  }

  extern "C" void thrust_sort_96b(my_dev::dev_mem<uint4> &srcKeys, 
                                  my_dev::dev_mem<uint4> &sortedKeys,
                                  my_dev::dev_mem<uint>  &temp_buffer,
                                  my_dev::dev_mem<uint>  &permutation_buffer,
                                  int N)
  {

      // wrap raw pointer with a device_ptr 
      thrust::device_ptr<uint4> keys     = thrust::device_pointer_cast(srcKeys.raw_p());
      thrust::device_ptr<uint4> outKeys = thrust::device_pointer_cast(sortedKeys.raw_p());
      thrust::device_ptr<uint> temp = thrust::device_pointer_cast(temp_buffer.raw_p());
      thrust::device_ptr<uint> permutation = thrust::device_pointer_cast(permutation_buffer.raw_p());
      
      // initialize permutation to [0, 1, 2, ... ,N-1]
      thrust::sequence(permutation, permutation + N);

      // sort z, y, x
      update_permutation<2>(keys, permutation, temp, N);
      update_permutation<1>(keys, permutation, temp, N);
      update_permutation<0>(keys, permutation, temp, N);

      // Note: keys have not been modified
      // Note: permutation now maps unsorted keys to sorted order

      thrust::gather(permutation, permutation + N, keys, outKeys);
  }

#endif


__device__ int undilate3(uint2 key) {
  int x, value = 0;
  
  key.x = key.x & 0x09249249;
  key.y = key.y & 0x09249249;
  
  // undilate first 10 bits

  x = key.y & 0x3FFFF;
  x = ((x <<  4) + (x << 2) + x) & 0x0E070381;
  x = ((x << 12) + (x << 6) + x) & 0x0FF80001;
  x = ((x << 18) + x) & 0x0FFC0000;
  value = value | (x >> 18);
  
  x = (key.y >> 18) & 0x3FFFF;
  x = ((x <<  4) + (x << 2) + x) & 0x0E070381;
  x = ((x << 12) + (x << 6) + x) & 0x0FF80001;
  x = ((x << 18) + x) & 0x0FFC0000;
  value = value | (x >> 12);
  

  // undilate second 10 bits

  x = key.x & 0x3FFFF;
  x = ((x <<  4) + (x << 2) + x) & 0x0E070381;
  x = ((x << 12) + (x << 6) + x) & 0x0FF80001;
  x = ((x << 18) + x) & 0x0FFC0000;
  value = value | ((x >> 18) << 10);
  
  x = (key.x >> 18) & 0x3FFFF;
  x = ((x <<  4) + (x << 2) + x) & 0x0E070381;
  x = ((x << 12) + (x << 6) + x) & 0x0FF80001;
  x = ((x << 18) + x) & 0x0FFC0000;
  value = value | ((x >> 12) << 10);
  
  return value;
}


__device__ uint2 dilate3(int value) {
  unsigned int x;
  uint2 key;
  
  // dilate first 10 bits

  x = value & 0x03FF;
  x = ((x << 16) + x) & 0xFF0000FF;
  x = ((x <<  8) + x) & 0x0F00F00F;
  x = ((x <<  4) + x) & 0xC30C30C3;
  x = ((x <<  2) + x) & 0x49249249;
  key.y = x;

  // dilate second 10 bits

  x = (value >> 10) & 0x03FF;
  x = ((x << 16) + x) & 0xFF0000FF;
  x = ((x <<  8) + x) & 0x0F00F00F;
  x = ((x <<  4) + x) & 0xC30C30C3;
  x = ((x <<  2) + x) & 0x49249249;
  key.x = x;

  return key;
} 

#if 0
//Morton order
__device__ uint2 get_key(int4 crd) {
  uint2 key, key1;
  key  = dilate3(crd.x);

  key1 = dilate3(crd.y);
  key.x = key.x | (key1.x << 1);
  key.y = key.y | (key1.y << 1);

  key1 = dilate3(crd.z);
  key.x = key.x | (key1.x << 2);
  key.y = key.y | (key1.y << 2);

  return key;
}

#else

__device__ uint4 get_key(int4 crd)
{
  const int bits = 30;  //20 to make it same number as morton order
  int i,xi, yi, zi;
  int mask;
  int key;
    
  //0= 000, 1=001, 2=011, 3=010, 4=110, 5=111, 6=101, 7=100
  //000=0=0, 001=1=1, 011=3=2, 010=2=3, 110=6=4, 111=7=5, 101=5=6, 100=4=7
  const int C[8] = {0, 1, 7, 6, 3, 2, 4, 5};
    
  int temp;
    
  mask = 1 << (bits - 1);
  key  = 0;

  uint4 key_new;
    
  for(i = 0; i < bits; i++, mask >>= 1)
  {
    xi = (crd.x & mask) ? 1 : 0;
    yi = (crd.y & mask) ? 1 : 0;
    zi = (crd.z & mask) ? 1 : 0;        

    int index = (xi << 2) + (yi << 1) + zi;
      
    if(index == 0)
    {
      temp = crd.z; crd.z = crd.y; crd.y = temp;
    }
    else  if(index == 1 || index == 5)
    {
      temp = crd.x; crd.x = crd.y; crd.y = temp;
    }
    else  if(index == 4 || index == 6)
    {
      crd.x = (crd.x) ^ (-1);
      crd.z = (crd.z) ^ (-1);
    }
    else  if(index == 7 || index == 3)
    {
      temp = (crd.x) ^ (-1);         
      crd.x = (crd.y) ^ (-1);
      crd.y = temp;
    }
    else
    {
      temp = (crd.z) ^ (-1);         
      crd.z = (crd.y) ^ (-1);
      crd.y = temp;          
    }   

    key = (key << 3) + C[index];

    if(i == 19)
    {
      key_new.y = key;
      key = 0;
    }
    if(i == 9)
    {
      key_new.x = key;
      key = 0;
    }
  } //end for

   key_new.z = key;

  return key_new;
}

#endif


__device__ uint4 get_mask(int level) {
  int mask_levels = 3*max(MAXLEVELS - level, 0);
  uint4 mask = {0x3FFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF,0xFFFFFFFF};

  if (mask_levels > 60)
  {
    mask.z = 0;
    mask.y = 0;
    mask.x = (mask.x >> (mask_levels - 60)) << (mask_levels - 60);
  }
  else if (mask_levels > 30) {
    mask.z = 0;
    mask.y = (mask.y >> (mask_levels - 30)) << (mask_levels - 30);
  } else {
    mask.z = (mask.z >> mask_levels) << mask_levels;
  }

  return mask;
}

__device__ uint4 get_imask(uint4 mask) {
  return make_uint4(0x3FFFFFFF ^ mask.x, 0xFFFFFFFF ^ mask.y, 0xFFFFFFFF ^ mask.z, 0);
}



__device__ int4 get_crd(uint2 key) {
  int4 crd;

  crd.x = undilate3(key);
  crd.y = undilate3(make_uint2(key.x >> 1, key.y >> 1));
  crd.z = undilate3(make_uint2(key.x >> 2, key.y >> 2));
  
  return crd;
}

__device__ inline int cmp_uint2(uint2 a, uint2 b) {
  if      (a.x < b.x) return -1;
  else if (a.x > b.x) return +1;
  else {
    if       (a.y < b.y) return -1;
    else  if (a.y > b.y) return +1;
    return 0;
  }  
}

__device__ int cmp_uint4(uint4 a, uint4 b) {
  if      (a.x < b.x) return -1;
  else if (a.x > b.x) return +1;
  else {
    if       (a.y < b.y) return -1;
    else  if (a.y > b.y) return +1;
    else {
      if       (a.z < b.z) return -1;
      else  if (a.z > b.z) return +1;
      return 0;
    } //end z    
  }  //end y
} //end x, function


//Binary search of the key within certain bounds (cij.x, cij.y)
__device__ int find_key(uint4 key, uint2 cij, uint4 *keys) {
  int l = cij.x;
  int r = cij.y - 1;
  while (r - l > 1) {
    int m = (r + l) >> 1;
    int cmp = cmp_uint4(keys[m], key);
    if (cmp == -1) {
      l = m;
    } else { 
      r = m;
    }
  }
  if (cmp_uint4(keys[l], key) >= 0) return l;

  return r;
}



__device__ float2 ds_accumulate(float2 a, float b){
  float tmp = a.x + b;
  float del = (tmp - a.x) - b;
  a.x = tmp;
  a.y -= del;
  return a;
}
__device__ float2 ds_regularise(float2 a){
  float tmp = a.x + a.y;
  a.y -= (tmp - a.x);
  a.x = tmp;
  return a;
}


__device__ void sh_MinMax(int i, int j, float3 *r_min, float3 *r_max, 
                          volatile float3 *sh_rmin, volatile  float3 *sh_rmax)
{
  sh_rmin[i].x  = (*r_min).x = fminf((*r_min).x, sh_rmin[j].x);
  sh_rmin[i].y  = (*r_min).y = fminf((*r_min).y, sh_rmin[j].y);
  sh_rmin[i].z  = (*r_min).z = fminf((*r_min).z, sh_rmin[j].z);
  sh_rmax[i].x  = (*r_max).x = fmaxf((*r_max).x, sh_rmax[j].x);
  sh_rmax[i].y  = (*r_max).y = fmaxf((*r_max).y, sh_rmax[j].y);
  sh_rmax[i].z  = (*r_max).z = fmaxf((*r_max).z, sh_rmax[j].z);
}

__device__ void MinMaxPos(float4 pos, float4 &rmax, float4 &rmin)
{
      rmin.x  = fminf(pos.x, rmin.x);
      rmin.y  = fminf(pos.y, rmin.y);
      rmin.z  = fminf(pos.z, rmin.z);
      rmax.x  = fmaxf(pos.x, rmax.x); 
      rmax.y  = fmaxf(pos.y, rmax.y); 
      rmax.z  = fmaxf(pos.z, rmax.z); 
}


__device__ real4 get_pos(uint2 key, float size, float4 corner) {
  real4 pos;
  pos.w = size;
  
  int4 crd = get_crd(key);
  float domain_fac = corner.w;
  pos.x = crd.x*domain_fac + corner.x;
  pos.y = crd.y*domain_fac + corner.y;
  pos.z = crd.z*domain_fac + corner.z;

  return pos;
}

#if 0
/***
**** --> prefix calculation via Horn(2005) data-parallel algoritm
***/
#define BTEST(x) (-(int)(x))
template<int DIM2>
__device__ int calc_prefix(int N, int* prefix_in, int tid) {
  int x, y = 0;

  const int DIM = 1 << DIM2;
  
  for (int p = 0; p < N; p += DIM) {
    int *prefix = &prefix_in[p];

    x = prefix[tid -  1]; __syncthreads(); prefix[tid] += x & BTEST(tid >=  1); __syncthreads();
    x = prefix[tid -  2]; __syncthreads(); prefix[tid] += x & BTEST(tid >=  2); __syncthreads();
    x = prefix[tid -  4]; __syncthreads(); prefix[tid] += x & BTEST(tid >=  4); __syncthreads();
    x = prefix[tid -  8]; __syncthreads(); prefix[tid] += x & BTEST(tid >=  8); __syncthreads();
    x = prefix[tid - 16]; __syncthreads(); prefix[tid] += x & BTEST(tid >= 16); __syncthreads();
    if (DIM2 >= 6) {x = prefix[tid - 32]; __syncthreads(); prefix[tid] += x & BTEST(tid >= 32); __syncthreads();}
    if (DIM2 >= 7) {x = prefix[tid - 64]; __syncthreads(); prefix[tid] += x & BTEST(tid >= 64); __syncthreads();}
    if (DIM2 >= 8) {x = prefix[tid -128]; __syncthreads(); prefix[tid] += x & BTEST(tid >=128); __syncthreads();}
    

    prefix[tid] += y;
    __syncthreads();

    y = prefix[DIM-1];
    __syncthreads();
  }

  return y;
} 

template<int DIM2>
__device__ int calc_prefix(int* prefix, int tid, int value) {
  int  x;
  
  const int DIM = 1 << DIM2;

  prefix[tid] = value;
  __syncthreads();

#if 1
  x = prefix[tid -  1]; __syncthreads(); prefix[tid] += x & BTEST(tid >=  1); __syncthreads();
  x = prefix[tid -  2]; __syncthreads(); prefix[tid] += x & BTEST(tid >=  2); __syncthreads();
  x = prefix[tid -  4]; __syncthreads(); prefix[tid] += x & BTEST(tid >=  4); __syncthreads();
  x = prefix[tid -  8]; __syncthreads(); prefix[tid] += x & BTEST(tid >=  8); __syncthreads();
  x = prefix[tid - 16]; __syncthreads(); prefix[tid] += x & BTEST(tid >= 16); __syncthreads();
  if (DIM2 >= 6) {x = prefix[tid - 32]; __syncthreads(); prefix[tid] += x & BTEST(tid >= 32); __syncthreads();}
  if (DIM2 >= 7) {x = prefix[tid - 64]; __syncthreads(); prefix[tid] += x & BTEST(tid >= 64); __syncthreads();}
  if (DIM2 >= 8) {x = prefix[tid -128]; __syncthreads(); prefix[tid] += x & BTEST(tid >=128); __syncthreads();}

  x = prefix[DIM - 1];
  __syncthreads();
  return x;
#else
  
  int offset = 0;
  int tid2 = tid << 1;

#pragma unroll
  for (int d = DIM >> 1; d > 0; d >>= 1) {
    __syncthreads();

    int iflag = BTEST(tid < d);
    int ai = (((tid2 + 1) << offset) - 1) & iflag;
    int bi = (((tid2 + 2) << offset) - 1) & iflag;
    
    prefix[bi] += prefix[ai] & iflag;
    offset++;
  }

  // clear the last element
  if (tid == 0) prefix[DIM - 1] = 0;

  // traverse down the tree building the scan in place
#pragma unroll
  for (int d = 1; d < DIM; d <<= 1) {
    offset--;
    __syncthreads();
    
    int iflag = BTEST(tid < d);
    int ai = (((tid2 + 1) << offset) - 1) & iflag;
    int bi = (((tid2 + 2) << offset) - 1) & iflag;
    
    int t       = prefix[ai];
    if (tid < d) {
      prefix[ai]  = (prefix[bi] & iflag) + (t & BTEST(tid >= d));
      prefix[bi] += t & iflag;
    }
  }
  __syncthreads();

  prefix[tid] += value;
  __syncthreads();
  
  x = prefix[DIM - 1];
  __syncthreads();
  return x;
#endif
}
#endif


#endif
